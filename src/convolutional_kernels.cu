#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "batchnorm_layer.h"
#include "blas.h"
#include "box.h"
#include "col2im.h"
#include "convolutional_layer.h"
#include "dark_cuda.h"
#include "gemm.h"
#include "im2col.h"
#include "utils.h"

__global__ void add_bias_kernel(float* output, float* biases, int batch,
    int filters, int spatial, int current_size)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= current_size)
    return;

  int f = (index / spatial) % filters;
  output[index] += biases[f];
}

void add_bias_gpu(
    float* output, float* biases, int batch, int filters, int spatial)
{
  const int current_size = batch * filters * spatial;
  const int num_blocks = get_number_of_blocks(current_size, BLOCK);

  add_bias_kernel<<<num_blocks, BLOCK, 0, get_cuda_stream()>>>(
      output, biases, batch, filters, spatial, current_size);
  CHECK_CUDA(hipPeekAtLastError());
}

__global__ void backward_bias_kernel(
    float* bias_updates, float* delta, int batch, int n, int size)
{
  __shared__ float part[BLOCK];
  int i, b;
  int filter = blockIdx.x;
  int p = threadIdx.x;
  float sum = 0;
  for (b = 0; b < batch; ++b)
  {
    for (i = 0; i < size; i += BLOCK)
    {
      int index = p + i + size * (filter + n * b);
      sum += (p + i < size) ? delta[index] : 0;
    }
  }
  part[p] = sum;
  __syncthreads();
  if (p == 0)
  {
    for (i = 0; i < BLOCK; ++i) bias_updates[filter] += part[i];
  }
}

void backward_bias_gpu(
    float* bias_updates, float* delta, int batch, int n, int size)
{
  backward_bias_kernel<<<n, BLOCK, 0, get_cuda_stream()>>>(
      bias_updates, delta, batch, n, size);
  CHECK_CUDA(hipPeekAtLastError());
}

__global__ void binarize_kernel(float* x, int n, float* binary)
{
  int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (i >= n)
    return;
  binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float* x, int n, float* binary)
{
  binarize_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream()>>>(
      x, n, binary);
  CHECK_CUDA(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(
    float* input, int n, int size, float* binary)
{
  int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (s >= size)
    return;
  int i = 0;
  float mean = 0;
  for (i = 0; i < n; ++i)
  {
    mean += fabs(input[i * size + s]);
  }
  mean = mean / n;
  for (i = 0; i < n; ++i)
  {
    binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
  }
}

void binarize_input_gpu(float* input, int n, int size, float* binary)
{
  binarize_input_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream()>>>(
      input, n, size, binary);
  CHECK_CUDA(hipPeekAtLastError());
}

__global__ void binarize_weights_kernel(
    float* weights, int n, int size, float* binary)
{
  int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (f >= n)
    return;
  int i = 0;
  float mean = 0;
  for (i = 0; i < size; ++i)
  {
    mean += fabs(weights[f * size + i]);
  }
  mean = mean / size;
  for (i = 0; i < size; ++i)
  {
    binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    // binary[f*size + i] = weights[f*size + i];
  }
}

void binarize_weights_gpu(float* weights, int n, int size, float* binary)
{
  binarize_weights_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream()>>>(
      weights, n, size, binary);
  CHECK_CUDA(hipPeekAtLastError());
}

__global__ void set_zero_kernel(float* src, int size)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size)
    src[i] = 0;
}

__inline__ __device__ float warpAllReduceSum(float val)
{
  for (int mask = WARP_SIZE / 2; mask > 0; mask /= 2)
#if CUDART_VERSION >= 9000
    val += __shfl_xor_sync(0xffffffff, val, mask);
#else
    val += __shfl_xor(val, mask);
#endif
  return val;
}

// only if (size % 32 == 0)
__global__ void reduce_kernel(
    float* weights, int n, int size, float* mean_arr_gpu)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int f = i / size;
  if (f >= n)
    return;
  float warp_mean = warpAllReduceSum(fabs(weights[i]));
  if (i % 32 == 0)
    atomicAdd(&mean_arr_gpu[f], warp_mean / size);
}

__global__ void binarize_weights_mean_kernel(
    float* weights, int n, int size, float* binary, float* mean_arr_gpu)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int f = i / size;
  if (f >= n)
    return;
  float mean = mean_arr_gpu[f];
  binary[i] = (weights[i] > 0) ? mean : -mean;
}

void fast_binarize_weights_gpu(
    float* weights, int n, int size, float* binary, float* mean_arr_gpu)
{
  if (size % 32 == 0)
  {
    size_t gridsize = n * size;
    const int num_blocks =
        get_number_of_blocks(gridsize, BLOCK);  // gridsize / BLOCK + 1;

    set_zero_kernel<<<(n / BLOCK + 1), BLOCK, 0, get_cuda_stream()>>>(
        mean_arr_gpu, n);
    reduce_kernel<<<num_blocks, BLOCK, 0, get_cuda_stream()>>>(
        weights, n, size, mean_arr_gpu);
    binarize_weights_mean_kernel<<<num_blocks, BLOCK, 0, get_cuda_stream()>>>(
        weights, n, size, binary, mean_arr_gpu);
    CHECK_CUDA(hipPeekAtLastError());
  }
  else
  {
    binarize_weights_gpu(weights, n, size, binary);
  }
}

__global__ void cuda_f32_to_f16(float* input_f32, size_t size, half* output_f16)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size)
    output_f16[idx] = __float2half(input_f32[idx]);
  // if (idx < size) output_f16[idx] = __float2half_rn(input_f32[idx]); // can't
  // be compiled on Linux without casting
  // __float2half_ru, __float2half_rd, __float2half_rz, __float2half_rn
  // if (idx < size) *((unsigned short *)output_f16 + idx) =
  // __float2half(input_f32[idx]);
}

void cuda_convert_f32_to_f16(float* input_f32, size_t size, float* output_f16)
{
  cuda_f32_to_f16<<<get_number_of_blocks(size, BLOCK), BLOCK, 0,
      get_cuda_stream()>>>(input_f32, size, (half*)output_f16);
  CHECK_CUDA(hipPeekAtLastError());
}

__global__ void cuda_f16_to_f32(half* input_f16, size_t size, float* output_f32)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size)
    output_f32[idx] = __half2float(input_f16[idx]);
  // if (idx < size) output_f32[idx] = __half2float(*((unsigned short
  // *)input_f16 + idx));
}

void cuda_convert_f16_to_f32(float* input_f16, size_t size, float* output_f32)
{
  cuda_f16_to_f32<<<get_number_of_blocks(size, BLOCK), BLOCK, 0,
      get_cuda_stream()>>>((half*)input_f16, size, output_f32);
  CHECK_CUDA(hipPeekAtLastError());
}

half* cuda_make_f16_from_f32_array(float* src, size_t n)
{
  half* dst16;
  size_t size = sizeof(half) * n;
  CHECK_CUDA(hipMalloc((void**)&dst16, size));
  if (src)
  {
    assert(n > 0);
    cuda_convert_f32_to_f16(src, n, (float*)dst16);
  }
  if (!dst16)
    error("Cuda malloc failed\n");
  return dst16;
}

void ForwardConvolutionalLayerGpu(layer* l, NetworkState state)
{
  if (l->binary)
  {
    binarize_weights_gpu(l->weights_gpu, l->n,
        (l->c / l->groups) * l->size * l->size, l->binary_weights_gpu);
    swap_binary(l);
  }

  if (l->xnor)
  {
    if (!l->align_bit_weights_gpu || state.train)
    {
      fast_binarize_weights_gpu(l->weights_gpu, l->n,
          (l->c / l->groups) * l->size * l->size, l->binary_weights_gpu,
          l->mean_arr_gpu);
    }

    if (l->align_bit_weights_gpu && !state.train && l->c >= 32 &&
        l->stride_x == l->stride_y)
    {
      int m = l->n / l->groups;
      int k = l->size * l->size * l->c / l->groups;
      int n = l->out_w * l->out_h;
      int ldb_align = l->lda_align;
      size_t new_ldb = k + (ldb_align - k % ldb_align);  // (k / 8 + 1) * 8;

      if (l->c % 32 == 0)
      {
        int ldb_align = l->lda_align;
        size_t new_ldb = k + (ldb_align - k % ldb_align);  // (k / 8 + 1) * 8;
        int const new_c = l->c / 32;

        repack_input_gpu_bin(
            state.input, (uint32_t*)l->align_workspace_gpu, l->w, l->h, l->c);

        im2col_ongpu(l->align_workspace_gpu, new_c, l->h, l->w, l->size,
            l->stride, l->pad, state.workspace);

        int new_k = l->size * l->size * l->c / 32;

        transpose_uint32_gpu((uint32_t*)state.workspace,
            (uint32_t*)l->transposed_align_workspace_gpu, new_k, n, n, new_ldb);

        gemm_nn_custom_bin_mean_transposed_gpu(m, n, k,
            (unsigned char*)l->align_bit_weights_gpu, new_ldb,
            (unsigned char*)l->transposed_align_workspace_gpu, new_ldb,
            l->output_gpu, n, l->mean_arr_gpu, l->biases_gpu,
            l->activation == LEAKY, l->bin_conv_shortcut_in_gpu,
            l->bin_conv_shortcut_out_gpu);
      }
      else
      {
        im2col_align_ongpu(state.input, l->c, l->h, l->w, l->size, l->stride,
            l->pad, l->align_workspace_gpu, l->bit_align);

        float_to_bit_gpu(l->align_workspace_gpu,
            (unsigned char*)state.workspace, l->align_workspace_size);

        transpose_bin_gpu((unsigned char*)state.workspace,
            (unsigned char*)l->transposed_align_workspace_gpu, k, n,
            l->bit_align, new_ldb, 8);

        gemm_nn_custom_bin_mean_transposed_gpu(m, n, k,
            (unsigned char*)l->align_bit_weights_gpu, new_ldb,
            (unsigned char*)l->transposed_align_workspace_gpu, new_ldb,
            l->output_gpu, n, l->mean_arr_gpu, l->biases_gpu,
            l->activation == LEAKY, l->bin_conv_shortcut_in_gpu,
            l->bin_conv_shortcut_out_gpu);
      }

      if (l->activation == SWISH)
        activate_array_swish_ongpu(l->output_gpu, l->outputs * l->batch,
            l->activation_input_gpu, l->output_gpu);
      else if (l->activation == MISH)
        activate_array_mish_ongpu(l->output_gpu, l->outputs * l->batch,
            l->activation_input_gpu, l->output_gpu);
      else if (l->activation == NORM_CHAN)
        activate_array_normalize_channels_ongpu(l->output_gpu,
            l->outputs * l->batch, l->batch, l->out_c, l->out_w * l->out_h,
            l->output_gpu);
      else if (l->activation == NORM_CHAN_SOFTMAX)
        activate_array_normalize_channels_softmax_ongpu(l->output_gpu,
            l->outputs * l->batch, l->batch, l->out_c, l->out_w * l->out_h,
            l->output_gpu, 0);
      else if (l->activation == NORM_CHAN_SOFTMAX_MAXVAL)
        activate_array_normalize_channels_softmax_ongpu(l->output_gpu,
            l->outputs * l->batch, l->batch, l->out_c, l->out_w * l->out_h,
            l->output_gpu, 1);
      else if (l->activation != LINEAR && l->activation != LEAKY)
        activate_array_ongpu(
            l->output_gpu, l->outputs * l->batch, l->activation);

      return;
    }
  }

  if (l->xnor)
  {
    swap_binary(l);
    binarize_gpu(
        state.input, l->c * l->h * l->w * l->batch, l->binary_input_gpu);
    state.input = l->binary_input_gpu;
  }

#ifdef CUDNN
  float alpha = 1, beta = 0;

  int iteration_num = GetCurrIter(state.net);
  if (state.index != 0 && state.net->cudnn_half && !l->xnor &&
      (!state.train || (iteration_num > 3 * state.net->burn_in) &&
                           state.net->loss_scale != 1) &&
      (l->c / l->groups) % 8 == 0 && l->n % 8 == 0 && l->groups <= 1 &&
      l->size > 1)
  {
    // Note: For improved performance it is advised to use beta[0] = 0.0.
    // For Tensor Core: hipdnnSetConvolutionMathType() where hipdnnMathType_t
    // mathType = HIPDNN_TENSOR_OP_MATH;
    // 1. or HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM and use
    // HIPDNN_DATA_HALF
    // 2. or HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED
    // More:
    // http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#tensor_ops

    const size_t input16_size = l->batch * l->c * l->w * l->h;
    const size_t output16_size = l->batch * l->out_c * l->out_h * l->out_w;

    if (*state.net->max_input16_size < input16_size)
    {
      *state.net->max_input16_size = input16_size;
      if (*state.net->input16_gpu)
        cuda_free(*state.net->input16_gpu);
      assert(*state.net->max_input16_size > 0);
      *state.net->input16_gpu = (float*)cuda_make_f16_from_f32_array(
          NULL, *state.net->max_input16_size);
    }
    float* input16 = *state.net->input16_gpu;

    if (*state.net->max_output16_size < output16_size)
    {
      *state.net->max_output16_size = output16_size;
      if (*state.net->output16_gpu)
        cuda_free(*state.net->output16_gpu);
      assert(*state.net->max_output16_size > 0);
      *state.net->output16_gpu = (float*)cuda_make_f16_from_f32_array(
          NULL, *state.net->max_output16_size);
    }
    float* output16 = *state.net->output16_gpu;

    assert(input16_size > 0);
    cuda_convert_f32_to_f16(state.input, input16_size, input16);

    CHECK_CUDNN(hipdnnConvolutionForward(cudnn_handle(), &alpha,
        l->srcTensorDesc16, input16, l->weightDesc16, l->weights_gpu16,
        l->convDesc, l->fw_algo16, state.workspace, l->workspace_size, &beta,
        l->dstTensorDesc16, output16));

    if (l->batch_normalize)
    {
      if (state.train)  // Training
      {
        simple_copy_ongpu(l->outputs * l->batch / 2, output16, l->x_gpu);
        float one = 1.0f;
        float zero = 0.0f;
        // Batch-normalization can still take FP16 inputs and outputs, saving
        // half the bandwidth compared to FP32, it's just that the statistics
        // and value adjustment should be done in FP32.
        CHECK_CUDNN(hipdnnBatchNormalizationForwardTraining(cudnn_handle(),
            HIPDNN_BATCHNORM_SPATIAL, &one, &zero, l->normDstTensorDescF16,
            l->x_gpu,  // input
            l->normDstTensorDescF16,
            output16,  // output
            l->normTensorDesc,
            l->scales_gpu,  // input
            l->biases_gpu,  // input
            .01,
            l->rolling_mean_gpu,      // input/output (should be FP32)
            l->rolling_variance_gpu,  // input/output (should be FP32)
            .00001,
            l->mean_gpu,  // output (should be FP32) - optional cache to speedup
                          // hipdnnBatchNormalizationBackward()
            l->variance_gpu));  // output (should be FP32) - optional cache to
                                // speedup hipdnnBatchNormalizationBackward()

        cuda_convert_f16_to_f32(output16, output16_size, l->output_gpu);
        // forward_batchnorm_layer_gpu(l, state);
      }
      else  // Detection
      {
        cuda_convert_f16_to_f32(output16, output16_size, l->output_gpu);
        normalize_gpu(l->output_gpu, l->rolling_mean_gpu,
            l->rolling_variance_gpu, l->batch, l->out_c, l->out_h * l->out_w);
        scale_bias_gpu(l->output_gpu, l->scales_gpu, l->batch, l->out_c,
            l->out_h * l->out_w);
        add_bias_gpu(l->output_gpu, l->biases_gpu, l->batch, l->out_c,
            l->out_w * l->out_h);
      }
    }
    else  // BIAS only
    {
      cuda_convert_f16_to_f32(output16, output16_size, l->output_gpu);
      add_bias_gpu(
          l->output_gpu, l->biases_gpu, l->batch, l->n, l->out_w * l->out_h);
    }
  }
  else
  {
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn_handle(), &alpha,
        l->srcTensorDesc, state.input, l->weightDesc, l->weights_gpu,
        l->convDesc, l->fw_algo, state.workspace, l->workspace_size, &beta,
        l->dstTensorDesc, l->output_gpu));

    if (l->batch_normalize)
      ForwardBatchnormLayerGpu(l, state);
    else
      add_bias_gpu(
          l->output_gpu, l->biases_gpu, l->batch, l->n, l->out_w * l->out_h);
  }

#else
  fill_ongpu(l->outputs * l->batch, 0, l->output_gpu, 1);

  int i, j;
  int m = l->n / l->groups;
  int k = l->size * l->size * l->c / l->groups;
  int n = l->out_w * l->out_h;
  for (i = 0; i < l->batch; ++i)
  {
    for (j = 0; j < l->groups; ++j)
    {
      float* im =
          state.input + (i * l->groups + j) * l->c / l->groups * l->h * l->w;
      float* a = l->weights_gpu + j * l->nweights / l->groups;
      float* b = state.workspace;
      float* c = l->output_gpu + (i * l->groups + j) * n * m;
      if (l->size == 1)
      {
        b = im;
      }
      else
      {
        im2col_gpu_ext(im,                               // input
            l->c / l->groups,                            // input channels
            l->h, l->w,                                  // input size (h, w)
            l->size, l->size,                            // kernel size (h, w)
            l->pad * l->dilation, l->pad * l->dilation,  // padding (h, w)
            l->stride_y, l->stride_x,                    // stride (h, w)
            l->dilation, l->dilation,                    // dilation (h, w)
            state.workspace);                            // output
      }
      gemm_ongpu(0, 0, m, n, k, 1, a, k, b, n, 1, c, n);
    }
  }

  if (l->batch_normalize)
    ForwardBatchnormLayerGpu(l, state);
  else
    add_bias_gpu(
        l->output_gpu, l->biases_gpu, l->batch, l->n, l->out_w * l->out_h);
#endif

  if (l->activation == SWISH)
    activate_array_swish_ongpu(l->output_gpu, l->outputs * l->batch,
        l->activation_input_gpu, l->output_gpu);
  else if (l->activation == MISH)
    activate_array_mish_ongpu(l->output_gpu, l->outputs * l->batch,
        l->activation_input_gpu, l->output_gpu);
  else if (l->activation == NORM_CHAN)
    activate_array_normalize_channels_ongpu(l->output_gpu,
        l->outputs * l->batch, l->batch, l->out_c, l->out_w * l->out_h,
        l->output_gpu);
  else if (l->activation == NORM_CHAN_SOFTMAX)
    activate_array_normalize_channels_softmax_ongpu(l->output_gpu,
        l->outputs * l->batch, l->batch, l->out_c, l->out_w * l->out_h,
        l->output_gpu, 0);
  else if (l->activation == NORM_CHAN_SOFTMAX_MAXVAL)
    activate_array_normalize_channels_softmax_ongpu(l->output_gpu,
        l->outputs * l->batch, l->batch, l->out_c, l->out_w * l->out_h,
        l->output_gpu, 1);
  else if (l->activation != LINEAR)
    activate_array_ongpu(l->output_gpu, l->outputs * l->batch, l->activation);

  if (l->binary || l->xnor)
    swap_binary(l);

  if (l->antialiasing)
  {
    NetworkState s = {0};
    s.train = state.train;
    s.workspace = state.workspace;
    s.net = state.net;
    if (!state.train)
      s.index = state.index;  // don't use TC for training (especially without
                              // cuda_convert_f32_to_f16() )
    s.input = l->output_gpu;
    ForwardConvolutionalLayerGpu(l->input_layer, s);
    simple_copy_ongpu(
        l->outputs * l->batch, l->output_gpu, l->input_antialiasing_gpu);
    simple_copy_ongpu(l->input_layer->outputs * l->input_layer->batch,
        l->input_layer->output_gpu, l->output_gpu);
  }
}

void BackwardConvolutionalLayerGpu(layer* l, NetworkState state)
{
  if (l->antialiasing)
  {
    NetworkState s = {0};
    s.train = state.train;
    s.workspace = state.workspace;
    s.net = state.net;
    s.delta = l->delta_gpu;
    s.input = l->input_antialiasing_gpu;

    simple_copy_ongpu(l->input_layer->outputs * l->input_layer->batch,
        l->delta_gpu, l->input_layer->delta_gpu);
    BackwardConvolutionalLayerGpu(l->input_layer, s);

    simple_copy_ongpu(
        l->outputs * l->batch, l->input_antialiasing_gpu, l->output_gpu);
  }

  if (l->activation == SWISH)
    gradient_array_swish_ongpu(l->output_gpu, l->outputs * l->batch,
        l->activation_input_gpu, l->delta_gpu);
  else if (l->activation == MISH)
    gradient_array_mish_ongpu(
        l->outputs * l->batch, l->activation_input_gpu, l->delta_gpu);
  else if (l->activation == NORM_CHAN_SOFTMAX ||
           l->activation == NORM_CHAN_SOFTMAX_MAXVAL)
    gradient_array_normalize_channels_softmax_ongpu(l->output_gpu,
        l->outputs * l->batch, l->batch, l->out_c, l->out_w * l->out_h,
        l->delta_gpu);
  else if (l->activation == NORM_CHAN)
    gradient_array_normalize_channels_ongpu(l->output_gpu,
        l->outputs * l->batch, l->batch, l->out_c, l->out_w * l->out_h,
        l->delta_gpu);
  else
    gradient_array_ongpu(
        l->output_gpu, l->outputs * l->batch, l->activation, l->delta_gpu);

  if (!l->batch_normalize)
    backward_bias_gpu(
        l->bias_updates_gpu, l->delta_gpu, l->batch, l->n, l->out_w * l->out_h);

  float* original_input = state.input;

  if (l->xnor)
    state.input = l->binary_input_gpu;
#ifdef CUDNN
  float one = 1.f;
  float alpha = 1, beta = 0;

  int iteration_num = GetCurrIter(state.net);
  if (state.index != 0 && state.net->cudnn_half && !l->xnor &&
      (!state.train || (iteration_num > 3 * state.net->burn_in) &&
                           state.net->loss_scale != 1) &&
      (l->c / l->groups) % 8 == 0 && l->n % 8 == 0 && l->groups <= 1 &&
      l->size > 1)
  {
    const size_t input16_size = l->batch * l->c * l->w * l->h;
    const size_t delta16_size = l->batch * l->n * l->out_w * l->out_h;

    if (*state.net->max_input16_size < input16_size)
    {
      *state.net->max_input16_size = input16_size;
      if (*state.net->input16_gpu)
        cuda_free(*state.net->input16_gpu);
      assert(*state.net->max_input16_size > 0);
      *state.net->input16_gpu = (float*)cuda_make_f16_from_f32_array(
          NULL, *state.net->max_input16_size);
    }
    float* input16 = *state.net->input16_gpu;

    if (*state.net->max_output16_size < delta16_size)
    {
      *state.net->max_output16_size = delta16_size;
      if (*state.net->output16_gpu)
        cuda_free(*state.net->output16_gpu);
      assert(*state.net->max_output16_size > 0);
      *state.net->output16_gpu = (float*)cuda_make_f16_from_f32_array(
          NULL, *state.net->max_output16_size);
    }
    float* delta16 = *state.net->output16_gpu;

    assert(input16_size > 0);
    assert(delta16_size > 0);
    cuda_convert_f32_to_f16(state.input, input16_size, input16);
    cuda_convert_f32_to_f16(l->delta_gpu, delta16_size, delta16);

    if (l->batch_normalize)
    {
      float one = 1.0f;
      float zero = 0.0f;
      CHECK_CUDNN(hipdnnBatchNormalizationBackward(cudnn_handle(),
          HIPDNN_BATCHNORM_SPATIAL, &one, &zero, &one, &one,
          l->normDstTensorDescF16,
          l->x_gpu,  // input (input in BN-forward-inference)
          l->normDstTensorDescF16,
          delta16,  // input
          l->normDstTensorDescF16,
          l->output_gpu,  // l->x_norm_gpu,            // output (new delta)
          l->normTensorDesc,
          l->scales_gpu,         // input (should be FP32)
          l->scale_updates_gpu,  // output (should be FP32)
          l->bias_updates_gpu,   // output (should be FP32)
          .00001,
          l->mean_gpu,        // input (should be FP32)
          l->variance_gpu));  // input (should be FP32)

      simple_copy_ongpu(l->outputs * l->batch / 2, l->output_gpu, delta16);
    }

    // convert input: state.input (x), l->delta_gpu (y) from fp32 to fp16
    // get output: l->weight_updates_gpu (dw) and convert it to fp32 (ONLY if it
    // is fp16)

    // calculate conv weight updates
    // Already: l->weight_updates_gpu = (l->weight_updates_gpu -
    // l->weight*decay*batch*subdivision)*momentum
    //   so we should copy f32 to f16, or compute: f16=(w_up - w*d*b*s)*m
    assert((l->nweights) > 0);
    cuda_convert_f32_to_f16(
        l->weight_updates_gpu, l->nweights, l->weight_updates_gpu16);

    if (!l->train_only_bn)
    {
      CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn_handle(), &one,
          l->srcTensorDesc16, input16, l->ddstTensorDesc16, delta16,
          l->convDesc, l->bf_algo16, state.workspace, l->workspace_size, &one,
          l->dweightDesc16, l->weight_updates_gpu16));

      cuda_convert_f16_to_f32(
          l->weight_updates_gpu16, l->nweights, l->weight_updates_gpu);
    }

    if (state.delta)
    {
      if (l->binary || l->xnor)
        swap_binary(l);

      // http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
      // calculate delta for the next layer
      // convert input: l->weights_gpu (w), l->delta_gpu (dy) from fp32 to fp16
      // get output: state.delta (dx) and convert it to fp32 (ONLY if it is
      // fp16)
      CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn_handle(), &alpha,
          l->weightDesc16, l->weights_gpu16, l->ddstTensorDesc16, delta16,
          l->convDesc, l->bd_algo16, state.workspace, l->workspace_size, &beta,
          l->dsrcTensorDesc16, input16));

      cuda_convert_f16_to_f32(input16, input16_size, state.delta);

      if (l->binary || l->xnor)
        swap_binary(l);
      if (l->xnor)
        gradient_array_ongpu(original_input, l->batch * l->c * l->h * l->w,
            HARDTAN, state.delta);
    }
  }
  else
  {
    if (l->batch_normalize)
    {
      BackwardBatchnormLayerGpu(l, state);
    }

    if (!l->train_only_bn)
    {
      // calculate conv weight updates
      // if used: beta=1 then loss decreases faster
      CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn_handle(), &one,
          l->srcTensorDesc, state.input, l->ddstTensorDesc, l->delta_gpu,
          l->convDesc, l->bf_algo, state.workspace, l->workspace_size, &one,
          l->dweightDesc, l->weight_updates_gpu));
    }

    if (state.delta)
    {
      if (l->binary || l->xnor)
        swap_binary(l);
      // http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
      // calculate delta for the next layer
      CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn_handle(), &one,
          l->weightDesc, l->weights_gpu, l->ddstTensorDesc, l->delta_gpu,
          l->convDesc, l->bd_algo, state.workspace, l->workspace_size, &one,
          l->dsrcTensorDesc, state.delta));

      if (l->binary || l->xnor)
        swap_binary(l);
      if (l->xnor)
        gradient_array_ongpu(original_input, l->batch * l->c * l->h * l->w,
            HARDTAN, state.delta);
    }
  }

#else  // CUDNN
  if (l->batch_normalize)
    BackwardBatchnormLayerGpu(l, state);

  int m = l->n / l->groups;
  int n = l->size * l->size * l->c / l->groups;
  int k = l->out_w * l->out_h;

  int i, j;
  for (i = 0; i < l->batch; ++i)
  {
    for (j = 0; j < l->groups; ++j)
    {
      float* a = l->delta_gpu + (i * l->groups + j) * m * k;
      float* b = state.workspace;
      float* c = l->weight_updates_gpu + j * l->nweights / l->groups;

      float* im =
          state.input + (i * l->groups + j) * l->c / l->groups * l->h * l->w;

      if (!l->train_only_bn)
      {
        im2col_gpu_ext(im,                               // input
            l->c / l->groups,                            // input channels
            l->h, l->w,                                  // input size (h, w)
            l->size, l->size,                            // kernel size (h, w)
            l->pad * l->dilation, l->pad * l->dilation,  // padding (h, w)
            l->stride_y, l->stride_x,                    // stride (h, w)
            l->dilation, l->dilation,                    // dilation (h, w)
            state.workspace);                            // output
        gemm_ongpu(0, 1, m, n, k, 1, a, k, b, k, 1, c, n);
      }

      if (state.delta)
      {
        if (l->binary || l->xnor)
          swap_binary(l);

        float* a = l->weights_gpu + j * l->nweights / l->groups;
        float* b = l->delta_gpu + (i * l->groups + j) * m * k;
        float* c = state.workspace;

        gemm_ongpu(1, 0, n, k, m, 1, a, n, b, k, 0, c, k);

        float* delta =
            state.delta + (i * l->groups + j) * l->c / l->groups * l->h * l->w;

        col2im_gpu_ext(state.workspace,                  // input
            l->c / l->groups,                            // input channels
            l->h, l->w,                                  // input size (h, w)
            l->size, l->size,                            // kernel size (h, w)
            l->pad * l->dilation, l->pad * l->dilation,  // padding size (h, w)
            l->stride_y, l->stride_x,                    // stride size (h, w)
            l->dilation, l->dilation,                    // dilation size (h, w)
            delta);                                      // output (delta)

        if (l->binary || l->xnor)
          swap_binary(l);

        if (l->xnor)
          gradient_array_ongpu(original_input + i * l->c * l->h * l->w,
              l->c * l->h * l->w, HARDTAN,
              state.delta + i * l->c * l->h * l->w);
      }
    }
  }
#endif
}

void PullConvolutionalLayer(layer* l)
{
  cuda_pull_array_async(l->weights_gpu, l->weights, l->nweights);
  cuda_pull_array_async(l->biases_gpu, l->biases, l->n);
  cuda_pull_array_async(l->weight_updates_gpu, l->weight_updates, l->nweights);
  cuda_pull_array_async(l->bias_updates_gpu, l->bias_updates, l->n);
  if (l->batch_normalize)
  {
    cuda_pull_array_async(l->scales_gpu, l->scales, l->n);
    cuda_pull_array_async(l->rolling_mean_gpu, l->rolling_mean, l->n);
    cuda_pull_array_async(l->rolling_variance_gpu, l->rolling_variance, l->n);
  }
  if (l->adam)
  {
    cuda_pull_array_async(l->m_gpu, l->m, l->nweights);
    cuda_pull_array_async(l->v_gpu, l->v, l->nweights);
  }
  CHECK_CUDA(hipPeekAtLastError());
  hipStreamSynchronize(get_cuda_stream());
}

void PushConvolutionalLayer(layer* l)
{
  cuda_push_array(l->weights_gpu, l->weights, l->nweights);
#ifdef CUDNN_HALF
  assert(l->nweights > 0);
  cuda_convert_f32_to_f16(l->weights_gpu, l->nweights, l->weights_gpu16);
#endif
  cuda_push_array(l->biases_gpu, l->biases, l->n);
  if (l->train)
  {
    cuda_push_array(l->weight_updates_gpu, l->weight_updates, l->nweights);
    cuda_push_array(l->bias_updates_gpu, l->bias_updates, l->n);
  }
  if (l->batch_normalize)
  {
    cuda_push_array(l->scales_gpu, l->scales, l->n);
    cuda_push_array(l->rolling_mean_gpu, l->rolling_mean, l->n);
    cuda_push_array(l->rolling_variance_gpu, l->rolling_variance, l->n);
  }
  if (l->adam)
  {
    cuda_push_array(l->m_gpu, l->m, l->nweights);
    cuda_push_array(l->v_gpu, l->v, l->nweights);
  }
  CHECK_CUDA(hipPeekAtLastError());
}

void UpdateConvolutionalLayerGpu(layer* l, int batch, float learning_rate_init,
    float momentum, float decay, float loss_scale)
{
  float learning_rate = learning_rate_init * l->learning_rate_scale;

  // Loss scale for Mixed-Precision on Tensor-Cores
  if (loss_scale != 1.0)
  {
    if (l->weight_updates_gpu && l->nweights > 0)
      scal_ongpu(l->nweights, 1.0 / loss_scale, l->weight_updates_gpu, 1);
    if (l->bias_updates_gpu && l->n > 0)
      scal_ongpu(l->n, 1.0 / loss_scale, l->bias_updates_gpu, 1);
    if (l->scale_updates_gpu && l->n > 0)
      scal_ongpu(l->n, 1.0 / loss_scale, l->scale_updates_gpu, 1);
  }

  reset_nan_and_inf(l->weight_updates_gpu, l->nweights);
  fix_nan_and_inf(l->weights_gpu, l->nweights);

  if (l->adam)
  {
    adam_update_gpu(l->weights_gpu, l->weight_updates_gpu, l->m_gpu, l->v_gpu,
        l->B1, l->B2, l->eps, decay, learning_rate, l->nweights, batch, l->t);

    adam_update_gpu(l->biases_gpu, l->bias_updates_gpu, l->bias_m_gpu,
        l->bias_v_gpu, l->B1, l->B2, l->eps, decay, learning_rate, l->n, batch,
        l->t);
    if (l->scales_gpu)
    {
      adam_update_gpu(l->scales_gpu, l->scale_updates_gpu, l->scale_m_gpu,
          l->scale_v_gpu, l->B1, l->B2, l->eps, decay, learning_rate, l->n,
          batch, l->t);
    }
  }
  else
  {
    axpy_ongpu(l->nweights, -decay * batch, l->weights_gpu, 1,
        l->weight_updates_gpu, 1);
    axpy_ongpu(l->nweights, learning_rate / batch, l->weight_updates_gpu, 1,
        l->weights_gpu, 1);
    scal_ongpu(l->nweights, momentum, l->weight_updates_gpu, 1);

    axpy_ongpu(
        l->n, learning_rate / batch, l->bias_updates_gpu, 1, l->biases_gpu, 1);
    scal_ongpu(l->n, momentum, l->bias_updates_gpu, 1);

    if (l->scales_gpu)
    {
      axpy_ongpu(l->n, learning_rate / batch, l->scale_updates_gpu, 1,
          l->scales_gpu, 1);
      scal_ongpu(l->n, momentum, l->scale_updates_gpu, 1);
    }
  }

  if (l->clip)
    constrain_ongpu(l->nweights, l->clip, l->weights_gpu, 1);
}
