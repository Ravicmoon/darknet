#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "avgpool_layer.h"
#include "dark_cuda.h"

__global__ void forward_avgpool_layer_kernel(
    int n, int w, int h, int c, float* input, float* output)
{
  int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (id >= n)
    return;

  int k = id % c;
  id /= c;
  int b = id;

  int i;
  int out_index = (k + c * b);
  output[out_index] = 0;
  for (i = 0; i < w * h; ++i)
  {
    int in_index = i + h * w * (k + b * c);
    output[out_index] += input[in_index];
  }
  output[out_index] /= w * h;
}

__global__ void backward_avgpool_layer_kernel(
    int n, int w, int h, int c, float* in_delta, float* out_delta)
{
  int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
  if (id >= n)
    return;

  int k = id % c;
  id /= c;
  int b = id;

  int i;
  int out_index = (k + c * b);
  for (i = 0; i < w * h; ++i)
  {
    int in_index = i + h * w * (k + b * c);
    in_delta[in_index] += out_delta[out_index] / (w * h);
  }
}

void ForwardAvgpoolLayerGpu(layer* l, NetworkState state)
{
  size_t n = l->c * l->batch;

  forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0,
      get_cuda_stream()>>>(n, l->w, l->h, l->c, state.input, l->output_gpu);
  CHECK_CUDA(hipPeekAtLastError());
}

void BackwardAvgpoolLayerGpu(layer* l, NetworkState state)
{
  size_t n = l->c * l->batch;

  backward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0,
      get_cuda_stream()>>>(n, l->w, l->h, l->c, state.delta, l->delta_gpu);
  CHECK_CUDA(hipPeekAtLastError());
}
